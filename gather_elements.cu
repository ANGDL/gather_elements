#include "hip/hip_runtime.h"
//
// Created by ang on 2021/1/11.
//
#include <cassert>
#include <cmath>

# include "gather_elements.cuh"

#define KERNEL_BLOCK 1024

// cuda_gridsize
static
dim3 cuda_gridsize(unsigned int n, unsigned int blocks) {
    unsigned int k = (n - 1) / blocks + 1;
    unsigned int x = k;
    unsigned int y = 1;
    if (x > 65535) {
        x = static_cast<unsigned int>(ceil(sqrt((float) k)));
        y = (n - 1) / (x * blocks) + 1;
    }
    dim3 d = {x, y, 1};
    //printf("%ld %ld %ld %ld\n", n, x, y, x*y*KERNEL_BLOCK);
    return d;
}


__global__
void gather_elements_kernel(
        const float* input, const unsigned long* index, float* output, const unsigned int axis,
        unsigned int in_c, unsigned int in_h, unsigned int in_w,
        unsigned int idx_c, unsigned int idx_h, unsigned int idx_w){

    unsigned int out_idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (out_idx >= idx_c * idx_h * idx_w){
        return;
    }

    unsigned int i = out_idx / (idx_w * idx_h);
    unsigned int j = (out_idx - i * idx_w * idx_h) / idx_w;
    unsigned int k = out_idx - (i * idx_w * idx_h) - (j * idx_w);

//    printf("%u, %u %u %u\n", out_idx, i, j, k);

    unsigned int in_idx;

    if (0 == axis) {
        in_idx = index[out_idx] * in_h * in_w + j * in_w + k;
    }
    else if (1 == axis) {
        in_idx = i * in_h * in_w + index[out_idx] * in_w + k;
    }
    else{
        in_idx = i * in_h * in_w + j * in_w + index[out_idx];
    }

//    assert(out_idx < in_c * in_h * in_w);

//    printf("%u: %f\n", in_idx, input[in_idx]);

    output[out_idx] = input[in_idx];

//    printf("output: %f\n", output[out_idx]);
}


void gather_elements(
        const void* const* input,
        void** output,
        unsigned long axis,
        unsigned int in_c, unsigned int in_h, unsigned int in_w,
        unsigned int idx_c, unsigned int idx_h, unsigned int idx_w,
        hipStream_t stream){

    unsigned int data_size = idx_c * idx_h * idx_w;
    unsigned int blocks = KERNEL_BLOCK;

    if (KERNEL_BLOCK > data_size){
        blocks = data_size;
    }
    gather_elements_kernel<<<cuda_gridsize(data_size, blocks), blocks, 0, stream>>>(
            (float*)input[0], (unsigned long*)input[1], (float*)output[0], axis,
            in_c, in_h, in_w, idx_c, idx_h, idx_w);

    hipDeviceSynchronize();
}